#include "hip/hip_runtime.h"
/*
 * Copyright © Advanced Micro Devices, Inc. All rights reserved.
 * Adapted from
 * https://github.com/NVIDIA/TensorRT-LLM/blob/v0.7.1/cpp/tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.cu
 * Copyright (C) 2024-2025, The vLLM team.
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION & AFFILIATES. All rights
 * reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "dispatch_utils.h"
#include "hip_compat.h"
#include "hip_reduce.h"
#include "py_itfs_common.h"
#include "vec_convert.h"
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <torch/all.h>

#ifndef USE_ROCM
#include <hipcub/hipcub.hpp>
#include <cub/util_type.cuh>
#else
#include <hipcub/hipcub.hpp>
#include <hipcub/util_type.hpp>
#endif

#define MAX(a, b) ((a) > (b) ? (a) : (b))
#define MIN(a, b) ((a) < (b) ? (a) : (b))

namespace vllm {
namespace moe {

/// Aligned array type
template <typename T,
          /// Number of elements in the array
          int N,
          /// Alignment requirement in bytes
          int Alignment = sizeof(T) * N>
class alignas(Alignment) AlignedArray
{
    float data[N];
};

// ====================== Softmax things ===============================
// We have our own implementation of softmax here so we can support transposing the output
// in the softmax kernel when we extend this module to support expert-choice routing.
template <typename DTYPE, int TPB>
__launch_bounds__(TPB) __global__
    void moeSoftmax(const DTYPE* input, const bool* finished, float* output, const int num_cols)
{
    using BlockReduce = hipcub::BlockReduce<float, TPB>;
    __shared__ typename BlockReduce::TempStorage tmpStorage;

    __shared__ float normalizing_factor;
    __shared__ float float_max;

    const int thread_row_offset = blockIdx.x * num_cols;

    hipcub::Sum sum;
    float threadData(-FLT_MAX);

    // Don't touch finished rows.
    if((finished != nullptr) && finished[blockIdx.x])
    {
        return;
    }

    for(int ii = threadIdx.x; ii < num_cols; ii += TPB)
    {
        const int idx = thread_row_offset + ii;
        threadData    = max(static_cast<float>(input[idx]), threadData);
    }

    const float maxElem = BlockReduce(tmpStorage).Reduce(threadData, hipcub::Max());
    if(threadIdx.x == 0)
    {
        float_max = maxElem;
    }
    __syncthreads();

    threadData = 0;

    for(int ii = threadIdx.x; ii < num_cols; ii += TPB)
    {
        const int idx = thread_row_offset + ii;
        threadData += exp((static_cast<float>(input[idx]) - float_max));
    }

    const auto Z = BlockReduce(tmpStorage).Reduce(threadData, sum);

    if(threadIdx.x == 0)
    {
        normalizing_factor = 1.f / Z;
    }
    __syncthreads();

    for(int ii = threadIdx.x; ii < num_cols; ii += TPB)
    {
        const int idx   = thread_row_offset + ii;
        const float val = exp((static_cast<float>(input[idx]) - float_max)) * normalizing_factor;
        output[idx]     = val;
    }
}

template <int TPB>
__launch_bounds__(TPB) __global__ void moeTopK(const float* inputs_after_softmax,
                                               const bool* finished,
                                               float* output,
                                               int* indices,
                                               int* source_rows,
                                               const int num_experts,
                                               const int k,
                                               const int start_expert,
                                               const int end_expert,
                                               const bool need_renorm)
{

    using cub_kvp     = hipcub::KeyValuePair<int, float>;
    using BlockReduce = hipcub::BlockReduce<cub_kvp, TPB>;
    __shared__ typename BlockReduce::TempStorage tmpStorage;

    cub_kvp thread_kvp;
    hipcub::ArgMax arg_max;

    const int num_rows  = gridDim.x;
    const int block_row = blockIdx.x;

    float renorm_value           = 0.0f;
    const bool row_is_active     = finished ? !finished[block_row] : true;
    const int thread_read_offset = blockIdx.x * num_experts;
    for(int k_idx = 0; k_idx < k; ++k_idx)
    {
        thread_kvp.key   = 0;
        thread_kvp.value = -1.f; // This is OK because inputs are probabilities

        cub_kvp inp_kvp;
        for(int expert = threadIdx.x; expert < num_experts; expert += TPB)
        {
            const int idx = thread_read_offset + expert;
            inp_kvp.key   = expert;
            inp_kvp.value = inputs_after_softmax[idx];

            for(int prior_k = 0; prior_k < k_idx; ++prior_k)
            {
                const int prior_winning_expert = indices[k * block_row + prior_k];

                if(prior_winning_expert == expert)
                {
                    inp_kvp = thread_kvp;
                }
            }

            thread_kvp = arg_max(inp_kvp, thread_kvp);
        }

        const cub_kvp result_kvp = BlockReduce(tmpStorage).Reduce(thread_kvp, arg_max);
        if(threadIdx.x == 0)
        {
            // Ignore experts the node isn't responsible for with expert parallelism
            const int expert              = result_kvp.key;
            const bool node_uses_expert   = expert >= start_expert && expert < end_expert;
            const bool should_process_row = row_is_active && node_uses_expert;

            const int idx = k * block_row + k_idx;
            output[idx]   = result_kvp.value;
            indices[idx]  = should_process_row ? (expert - start_expert) : num_experts;
            assert(indices[idx] >= 0);
            source_rows[idx] = k_idx * num_rows + block_row;

            if(need_renorm)
            {
                renorm_value += result_kvp.value;
            }
        }
        __syncthreads();
    }

    if(need_renorm && threadIdx.x == 0 && renorm_value != 0.f)
    {
        renorm_value = 1 / renorm_value;
        for(int k_idx = 0; k_idx < k; k_idx++)
        {
            int64_t const idx = k * block_row + k_idx;
            output[idx] *= renorm_value;
        }
    }
}

// ====================== TopK softmax things ===============================

/*
  A Top-K gating softmax written to exploit when the number of experts in the MoE layers
  are a small power of 2. This allows us to cleanly share the rows among the threads in
  a single warp and eliminate communication between warps (so no need to use shared mem).

  It fuses the softmax, max and argmax into a single kernel.

  Limitations:
  1) This implementation is intended for when the number of experts is a small power of 2.
  2) This implementation assumes k is small, but will work for any k.
*/

template <typename DTYPE,
          int VPT,
          int NUM_EXPERTS,
          int WARPS_PER_CTA,
          int BYTES_PER_LDG,
          bool need_renorm>
__launch_bounds__(WARPS_PER_CTA* WARP_SIZE) __global__
    void topkGatingSoftmax(const DTYPE* input,
                           const bool* finished,
                           float* output,
                           const int num_rows,
                           int* indices,
                           int* source_rows,
                           const int k,
                           const int start_expert,
                           const int end_expert,
                           const int output_stride,
                           const int indices_stride)
{
    // We begin by enforcing compile time assertions and setting up compile time constants.
    static_assert(VPT == (VPT & -VPT), "VPT must be power of 2");
    static_assert(NUM_EXPERTS == (NUM_EXPERTS & -NUM_EXPERTS), "NUM_EXPERTS must be power of 2");
    static_assert(BYTES_PER_LDG == (BYTES_PER_LDG & -BYTES_PER_LDG),
                  "BYTES_PER_LDG must be power of 2");
    static_assert(BYTES_PER_LDG <= 32, "BYTES_PER_LDG must be leq 32");

    // Number of bytes each thread pulls in per load
    static constexpr int ELTS_PER_LDG    = BYTES_PER_LDG / sizeof(DTYPE);
    static constexpr int ELTS_PER_ROW    = NUM_EXPERTS;
    static constexpr int THREADS_PER_ROW = ELTS_PER_ROW / VPT;
    static constexpr int LDG_PER_THREAD  = VPT / ELTS_PER_LDG;

    // Restrictions based on previous section.
    static_assert(VPT % ELTS_PER_LDG == 0,
                  "The elements per thread must be a multiple of the elements per ldg");
    static_assert(WARP_SIZE % THREADS_PER_ROW == 0,
                  "The threads per row must cleanly divide the threads per warp");
    static_assert(THREADS_PER_ROW == (THREADS_PER_ROW & -THREADS_PER_ROW),
                  "THREADS_PER_ROW must be power of 2");
    static_assert(THREADS_PER_ROW <= WARP_SIZE, "THREADS_PER_ROW can be at most warp size");

    // We have NUM_EXPERTS elements per row. We specialize for small #experts
    static constexpr int ELTS_PER_WARP = WARP_SIZE * VPT;
    static constexpr int ROWS_PER_WARP = ELTS_PER_WARP / ELTS_PER_ROW;
    static constexpr int ROWS_PER_CTA  = WARPS_PER_CTA * ROWS_PER_WARP;

    // Restrictions for previous section.
    static_assert(ELTS_PER_WARP % ELTS_PER_ROW == 0,
                  "The elts per row must cleanly divide the total elt per warp");

    // ===================== From this point, we finally start computing run-time variables.
    // ========================

    // Compute CTA and warp rows. We pack multiple rows into a single warp, and a block contains
    // WARPS_PER_CTA warps. This, each block processes a chunk of rows. We start by computing the
    // start row for each block.
    const int cta_base_row = blockIdx.x * ROWS_PER_CTA;

    // Now, using the base row per thread block, we compute the base row per warp.
    const int warp_base_row = cta_base_row + threadIdx.y * ROWS_PER_WARP;

    // The threads in a warp are split into sub-groups that will work on a row.
    // We compute row offset for each thread sub-group
    const int thread_row_in_warp = threadIdx.x / THREADS_PER_ROW;
    const int thread_row         = warp_base_row + thread_row_in_warp;

    // Threads with indices out of bounds should early exit here.
    if(thread_row >= num_rows)
    {
        return;
    }
    const bool row_is_active = finished ? !finished[thread_row] : true;

    // We finally start setting up the read pointers for each thread. First, each thread jumps to
    // the start of the row it will read.
    const DTYPE* thread_row_ptr = input + thread_row * ELTS_PER_ROW;

    // Now, we compute the group each thread belong to in order to determine the first column to
    // start loads.
    const int thread_group_idx         = threadIdx.x % THREADS_PER_ROW;
    const int first_elt_read_by_thread = thread_group_idx * ELTS_PER_LDG;
    const DTYPE* thread_read_ptr       = thread_row_ptr + first_elt_read_by_thread;

    // Determine the pointer type to use to read in the data depending on the BYTES_PER_LDG template
    // param. In theory, this can support all powers of 2 up to 16. NOTE(woosuk): The original
    // implementation uses CUTLASS aligned array here. We defined our own aligned array and use it
    // here to avoid the dependency on CUTLASS.
    using AccessType = ck_tile::vec_t<DTYPE, ELTS_PER_LDG>;
    using ChunkType  = ck_tile::vec_t<float, ELTS_PER_LDG>;
    using kvp        = hipcub::KeyValuePair<int, float>;
    // hipcub::ArgMax arg_max;
    // hipcub::ArgMin arg_min;

    // Finally, we pull in the data from global mem
    float row_chunk[VPT];
    ChunkType* row_chunk_vec_ptr          = reinterpret_cast<ChunkType*>(&row_chunk);
    const AccessType* vec_thread_read_ptr = reinterpret_cast<const AccessType*>(thread_read_ptr);
#pragma unroll
    for(int ii = 0; ii < LDG_PER_THREAD; ++ii)
    {
        row_chunk_vec_ptr[ii] = ck_tile::vec_convert<float, DTYPE, ELTS_PER_LDG>(
            vec_thread_read_ptr[ii * THREADS_PER_ROW]);
    }

    // First, do an in-thread max reduction to get the max value and its index.
    float thread_max      = row_chunk[0];
    int first_topk_expert = first_elt_read_by_thread;
#pragma unroll
    for(int ii = 1; ii < VPT; ++ii)
    {
        if(thread_max < row_chunk[ii])
        {
            thread_max        = row_chunk[ii];
            first_topk_expert = first_elt_read_by_thread + ii;
        }
    }

    // Now, we find the max within the thread group and distribute among the threads.
    auto arg_max = [](const kvp& a, const kvp& b) {
        if(a.value > b.value || (a.value == b.value && a.key < b.key))
        {
            return a;
        }
        return b;
    };
    kvp thread_kvp    = {first_topk_expert, thread_max};
    thread_kvp        = multithread_reduce(thread_kvp, arg_max, THREADS_PER_ROW);
    thread_max        = thread_kvp.value;
    first_topk_expert = thread_kvp.key;

    // From this point, thread max in all the threads have the max within the row.
    // Next: select top-K and compute softmax only on them; if need_renorm=false, normalize by the
    // full row.
    int start_col                           = first_elt_read_by_thread;
    static constexpr int COLS_PER_GROUP_LDG = ELTS_PER_LDG * THREADS_PER_ROW;

    float renorm_value = 0.0f;
    for(int k_idx = 0; k_idx < k; ++k_idx)
    {
        float max_val;
        int expert;
        if(k_idx == 0)
        {
            max_val = thread_max;
            expert  = first_topk_expert;
        }
        else
        {
            // First, each thread does the local argmax
            max_val = row_chunk[0];
            expert  = start_col;
#pragma unroll
            for(int ldg = 0, col = start_col; ldg < LDG_PER_THREAD;
                ++ldg, col += COLS_PER_GROUP_LDG)
            {
#pragma unroll
                for(int ii = 0; ii < ELTS_PER_LDG; ++ii)
                {
                    float val = row_chunk[ldg * ELTS_PER_LDG + ii];

                    // No check on the experts here since columns with the smallest index are
                    // processed first and only updated if > (not >=)
                    if(val > max_val)
                    {
                        max_val = val;
                        expert  = col + ii;
                    }
                }
            }

            // Now, we perform the argmax reduce.
            kvp thread_kvp = {expert, max_val};
            thread_kvp     = multithread_reduce(thread_kvp, arg_max, THREADS_PER_ROW);
            max_val        = thread_kvp.value;
            expert         = thread_kvp.key;
        }
        // Write the max for this k iteration to global memory.
        if(thread_group_idx == 0)
        {
            // Add a guard to ignore experts not included by this node
            const bool node_uses_expert   = expert >= start_expert && expert < end_expert;
            const bool should_process_row = row_is_active && node_uses_expert;

            // The lead thread from each sub-group will write out the final results to global
            // memory. (This will be a single) thread per row of the input/output matrices.
            const int output_idx  = output_stride * thread_row + k_idx;
            const int indices_idx = indices_stride * thread_row + k_idx;
            const int idx         = k * thread_row + k_idx;
            const float numer     = expf(max_val - thread_max);
            output[output_idx]    = numer;
            indices[indices_idx]  = should_process_row ? (expert - start_expert) : NUM_EXPERTS;
            source_rows[idx]      = k_idx * num_rows + thread_row;

            // Accumulate renorm scalar
            renorm_value += numer;
        }

        // Finally, we clear the value in the thread with the current max
        {
            const int ldg_group_for_expert     = expert / COLS_PER_GROUP_LDG;
            const int thread_to_clear_in_group = (expert / ELTS_PER_LDG) % THREADS_PER_ROW;

            // Only the thread in the group which produced the max will reset the "winning" value to
            // -inf.
            if(thread_group_idx == thread_to_clear_in_group)
            {
                const int offset_for_expert = expert % ELTS_PER_LDG;
                row_chunk[ldg_group_for_expert * ELTS_PER_LDG + offset_for_expert] = -INFINITY;
            }
        }
    }

    if constexpr(need_renorm)
    {
        if(thread_group_idx == 0 && renorm_value != 0.f)
        {
            renorm_value = 1 / renorm_value;
            for(int k_idx = 0; k_idx < k; k_idx++)
            {
                int64_t const idx = output_stride * thread_row + k_idx;
                output[idx] *= renorm_value;
            }
        }
    }
    else
    {
        float thread_sum_rest = 0.f;
#pragma unroll
        for(int ii = 0; ii < VPT; ++ii)
        {
            thread_sum_rest += expf(row_chunk[ii] - thread_max);
        }
        float row_sum_rest = multithread_reduce(
            thread_sum_rest, [](float a, float b) { return a + b; }, THREADS_PER_ROW);

        if(thread_group_idx == 0)
        {
            const float Z = renorm_value + row_sum_rest;
            if(Z != 0.f)
            {
                const float scale = 1.f / Z;
                for(int k_idx = 0; k_idx < k; ++k_idx)
                {
                    const int out_idx = output_stride * thread_row + k_idx;
                    output[out_idx] *= scale;
                }
            }
        }
    }
}

namespace detail {
// Constructs some constants needed to partition the work across threads at compile time.
template <typename DTYPE, int EXPERTS, int BYTES_PER_LDG>
struct TopkConstants
{
    static constexpr int ELTS_PER_LDG = BYTES_PER_LDG / sizeof(DTYPE);
    static_assert(EXPERTS / (ELTS_PER_LDG * WARP_SIZE) == 0 ||
                      EXPERTS % (ELTS_PER_LDG * WARP_SIZE) == 0,
                  "");
    static constexpr int VECs_PER_THREAD = MAX(1, EXPERTS / (ELTS_PER_LDG * WARP_SIZE));
    static constexpr int VPT             = VECs_PER_THREAD * ELTS_PER_LDG;
    static constexpr int THREADS_PER_ROW = EXPERTS / VPT;
    static constexpr int ROWS_PER_WARP   = WARP_SIZE / THREADS_PER_ROW;
};
} // namespace detail

template <typename DTYPE, int EXPERTS, int WARPS_PER_TB>
void topkGatingSoftmaxLauncherHelper(const DTYPE* input,
                                     const bool* finished,
                                     float* output,
                                     int* indices,
                                     int* source_row,
                                     const int num_rows,
                                     const int k,
                                     const int start_expert,
                                     const int end_expert,
                                     const int output_stride,
                                     const int indices_stride,
                                     const bool need_renorm,
                                     hipStream_t stream)
{
    static constexpr std::size_t MAX_BYTES_PER_LDG = 32;

    static constexpr int BYTES_PER_LDG = MIN(MAX_BYTES_PER_LDG, sizeof(DTYPE) * EXPERTS);
    using Constants                    = detail::TopkConstants<DTYPE, EXPERTS, BYTES_PER_LDG>;
    static constexpr int VPT           = Constants::VPT;
    static constexpr int ROWS_PER_WARP = Constants::ROWS_PER_WARP;
    const int num_warps                = (num_rows + ROWS_PER_WARP - 1) / ROWS_PER_WARP;
    const int num_blocks               = (num_warps + WARPS_PER_TB - 1) / WARPS_PER_TB;

    dim3 block_dim(WARP_SIZE, WARPS_PER_TB);
    if(need_renorm)
    {
        topkGatingSoftmax<DTYPE, VPT, EXPERTS, WARPS_PER_TB, BYTES_PER_LDG, true>
            <<<num_blocks, block_dim, 0, stream>>>(input,
                                                   finished,
                                                   output,
                                                   num_rows,
                                                   indices,
                                                   source_row,
                                                   k,
                                                   start_expert,
                                                   end_expert,
                                                   output_stride,
                                                   indices_stride);
    }
    else
    {
        topkGatingSoftmax<DTYPE, VPT, EXPERTS, WARPS_PER_TB, BYTES_PER_LDG, false>
            <<<num_blocks, block_dim, 0, stream>>>(input,
                                                   finished,
                                                   output,
                                                   num_rows,
                                                   indices,
                                                   source_row,
                                                   k,
                                                   start_expert,
                                                   end_expert,
                                                   output_stride,
                                                   indices_stride);
    }
}

#define LAUNCH_SOFTMAX(NUM_EXPERTS, WARPS_PER_TB)                                           \
    topkGatingSoftmaxLauncherHelper<DTYPE, NUM_EXPERTS, WARPS_PER_TB>(gating_output,        \
                                                                      nullptr,              \
                                                                      topk_weights,         \
                                                                      topk_indicies,        \
                                                                      token_expert_indices, \
                                                                      num_tokens,           \
                                                                      topk,                 \
                                                                      0,                    \
                                                                      num_experts,          \
                                                                      topk_weights_stride,  \
                                                                      topk_id_stride,       \
                                                                      need_renorm,          \
                                                                      stream);

template <typename DTYPE>
void topkGatingSoftmaxKernelLauncher(const DTYPE* gating_output,
                                     float* topk_weights,
                                     int* topk_indicies,
                                     int* token_expert_indices,
                                     float* softmax_workspace,
                                     const int num_tokens,
                                     const int num_experts,
                                     const int topk,
                                     const int topk_weights_stride,
                                     const int topk_id_stride,
                                     const bool need_renorm,
                                     hipStream_t stream)
{
    static constexpr int WARPS_PER_TB = 8;
    switch(num_experts)
    {
    case 1: LAUNCH_SOFTMAX(1, WARPS_PER_TB); break;
    case 2: LAUNCH_SOFTMAX(2, WARPS_PER_TB); break;
    case 4: LAUNCH_SOFTMAX(4, WARPS_PER_TB); break;
    case 8: LAUNCH_SOFTMAX(8, WARPS_PER_TB); break;
    case 16: LAUNCH_SOFTMAX(16, WARPS_PER_TB); break;
    case 32: LAUNCH_SOFTMAX(32, WARPS_PER_TB); break;
    case 64: LAUNCH_SOFTMAX(64, WARPS_PER_TB); break;
    case 128: LAUNCH_SOFTMAX(128, WARPS_PER_TB); break;
    case 256: LAUNCH_SOFTMAX(256, WARPS_PER_TB); break;
    case 512: LAUNCH_SOFTMAX(512, 2); break;
    default: {
        TORCH_CHECK(
            softmax_workspace != nullptr,
            "softmax_workspace must be provided for num_experts that are not a power of 2.");
        static constexpr int TPB = 256;
        moeSoftmax<DTYPE, TPB><<<num_tokens, TPB, 0, stream>>>(
            gating_output, nullptr, softmax_workspace, num_experts);
        moeTopK<TPB><<<num_tokens, TPB, 0, stream>>>(softmax_workspace,
                                                     nullptr,
                                                     topk_weights,
                                                     topk_indicies,
                                                     token_expert_indices,
                                                     num_experts,
                                                     topk,
                                                     0,
                                                     num_experts,
                                                     need_renorm);
    }
    }
}

template <typename scalar_t, int TOPK>
__global__ void moe_sum_kernel(scalar_t* __restrict__ out,         // [..., d]
                               const scalar_t* __restrict__ input, // [..., topk, d]
                               const int d)
{
    const int64_t token_idx = blockIdx.x;
    for(int64_t idx = threadIdx.x; idx < d; idx += blockDim.x)
    {
        scalar_t x = 0.0;
#pragma unroll
        for(int k = 0; k < TOPK; ++k)
        {
            x += VLLM_LDG(&input[token_idx * TOPK * d + k * d + idx]);
        }
        out[token_idx * d + idx] = x;
    }
}

} // namespace moe
} // namespace vllm

namespace aiter {

void topk_softmax(torch::Tensor& topk_weights,         // [num_tokens, topk]
                  torch::Tensor& topk_indices,         // [num_tokens, topk]
                  torch::Tensor& token_expert_indices, // [num_tokens, topk]
                  torch::Tensor& gating_output,        // [num_tokens, num_experts]
                  bool need_renorm)
{
    const int num_experts         = gating_output.size(-1);
    const int num_tokens          = gating_output.numel() / num_experts;
    const int topk                = topk_weights.size(-1);
    const int topk_weights_stride = topk_weights.stride(0);
    const int topk_id_stride      = topk_indices.stride(0);

    const bool is_pow_2          = (num_experts != 0) && ((num_experts & (num_experts - 1)) == 0);
    const bool needs_workspace   = !is_pow_2 || num_experts > 256;
    const int64_t workspace_size = needs_workspace ? num_tokens * num_experts : 0;

    const at::cuda::OptionalCUDAGuard device_guard(device_of(gating_output));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    torch::Tensor softmax_workspace =
        torch::empty({workspace_size}, gating_output.options().dtype(torch::kFloat32));
    VLLM_DISPATCH_FLOATING_TYPES(gating_output.scalar_type(), "topk_softmax", [&] {
        using input_dtype = typename t2ck<scalar_t>::type;
        vllm::moe::topkGatingSoftmaxKernelLauncher(
            reinterpret_cast<input_dtype*>(gating_output.data_ptr()),
            topk_weights.data_ptr<float>(),
            topk_indices.data_ptr<int>(),
            token_expert_indices.data_ptr<int>(),
            softmax_workspace.data_ptr<float>(),
            num_tokens,
            num_experts,
            topk,
            topk_weights_stride,
            topk_id_stride,
            need_renorm,
            stream);
    });
}

void moe_sum(torch::Tensor& input,  // [num_tokens, topk, hidden_size]
             torch::Tensor& output) // [num_tokens, hidden_size]
{
    const int hidden_size = input.size(-1);
    const int num_tokens  = output.numel() / hidden_size;
    const int topk        = input.size(1);

    dim3 grid(num_tokens);
    dim3 block(std::min(hidden_size, 1024));

    const at::cuda::OptionalCUDAGuard device_guard(device_of(output));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();

    switch(topk)
    {
    case 2:
        VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
            vllm::moe::moe_sum_kernel<scalar_t, 2><<<grid, block, 0, stream>>>(
                output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), hidden_size);
        });
        break;

    case 4:
        VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
            vllm::moe::moe_sum_kernel<scalar_t, 4><<<grid, block, 0, stream>>>(
                output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), hidden_size);
        });
        break;

    case 5:
        VLLM_DISPATCH_FLOATING_TYPES(input.scalar_type(), "moe_sum_kernel", [&] {
            vllm::moe::moe_sum_kernel<scalar_t, 5><<<grid, block, 0, stream>>>(
                output.data_ptr<scalar_t>(), input.data_ptr<scalar_t>(), hidden_size);
        });
        break;
    default: at::sum_out(output, input, 1); break;
    }
}

} // namespace aiter
